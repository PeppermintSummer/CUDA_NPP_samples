#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include "CUDA_color_conversion.h"

#include <npp.h>
#include <nppi.h>
#include <npps.h>

void YUV420pToRGB_NPP(const unsigned char *input, unsigned char *output, int width,
	int height, int device_id) {

	int curDev = -1;
	hipGetDevice(&curDev);
	if (curDev != device_id) {
		hipSetDevice(device_id);
	}

	Npp8u *pNppInput;
	int nppInputStep;
	pNppInput = nppiMalloc_8u_C1(width, height / 2 * 3, &nppInputStep);
	hipMemcpy(pNppInput, input, sizeof(Npp8u) * width * height / 2 * 3,
		hipMemcpyHostToDevice);

	Npp8u *pNppInput_V_half = pNppInput + width * height;

	Npp8u *pNppInput_U_half = pNppInput_V_half + width * height / 4;

	Npp8u *pNppInputArray[3] = { pNppInput, pNppInput_U_half, pNppInput_V_half };
	int nppInputSteps[3] = { width, width / 2, width / 2 };

	Npp8u *pNppOutput;
	int nppOutputStep;
	pNppOutput = nppiMalloc_8u_C3(width, height, &nppOutputStep);

	NppiSize nppSize;
	nppSize.width = width;
	nppSize.height = height;
	nppiYUV420ToBGR_8u_P3C3R(pNppInputArray, nppInputSteps, pNppOutput, width * 3,
		nppSize);

	hipMemcpy(output, pNppOutput, sizeof(unsigned char) * width * height * 3,
		hipMemcpyDeviceToHost);

	hipFree(pNppInput);
	hipFree(pNppOutput);

	if (curDev != device_id) {
		hipSetDevice(curDev);
	}
}

__global__ void cvtNV12_BGR(unsigned char* A,unsigned char* B,
	const int height,const int width,const int linesize)
{
	int IDX = blockDim.x * blockIdx.x + threadIdx.x;
	long len = width * height;
	if (IDX < len)
	{
		int j = IDX % width;
		int i = (IDX - j) / width;

		int bgr[3];
		int yIdx, uvIdx, idx;
		int y, u, v;

		yIdx = i * linesize + j;
		uvIdx = linesize * height + (i / 2)*linesize + j - j % 2;

		y = A[yIdx];
		u = A[uvIdx];
		v = A[uvIdx + 1];

		bgr[0] = y + 1.772 * (u - 128);
		bgr[1] = y - 0.34414 * (u - 128) - 0.71414 * (v - 128);
		bgr[2] = y + 1.402 * (v - 128);

		for (int k = 0; k < 3; k++) {
			idx = (i * width + j) * 3 + k;
			if (bgr[k] >= 0 && bgr[k] < 255) {
				B[idx] = bgr[k];
			}
			else {
				B[idx] = bgr[k] < 0 ? 0 : 255;
			}
		}
	}
}

int cvtColor(unsigned char *d_req,
	unsigned char *d_res,
	int resolution,
	int height,
	int width,
	int linesize)
{
	int threadPerBlock = 256;
	int blockPerGrid = (resolution + threadPerBlock - 1) / threadPerBlock;
	//params_type-->switch?
	cvtNV12_BGR << <blockPerGrid, threadPerBlock >> > (d_req, d_res, height, width, line);

	return 0;
}